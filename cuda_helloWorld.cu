
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void print_kernel(){
	printf("Block numarasi %d\t is parcacigi numarasi %d\n",blockIdx.x,threadIdx.x);
}

int main(){
	print_kernel<<<5,3>>>();
	hipDeviceSynchronize();
}
